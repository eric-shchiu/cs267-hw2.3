#include "hip/hip_runtime.h"
#include "common.h"
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/scan.h>

#define NUM_THREADS 256

// Global variable (host-side) for the number of blocks
int blks;

// __device__ variables (accessible by all kernels)
__device__ double bin_size;
__device__ int num_bins_x, num_bins_y;

int* d_bin_indices = nullptr;
int* d_bin_counts = nullptr;
int* d_bin_scan = nullptr;
int* d_particle_bins = nullptr;

// Kernel to assign particles to bins and count particles per bin
__global__ void assign_bins_gpu(particle_t* particles, int num_parts, int* bin_indices, int* bin_counts) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= num_parts) return;

    // Calculate bin coordinates using __device__ variables
    int bin_x = static_cast<int>(particles[tid].x / bin_size);
    int bin_y = static_cast<int>(particles[tid].y / bin_size);

    // Clamp bin indices to the valid range
    bin_x = max(0, min(bin_x, num_bins_x - 1));
    bin_y = max(0, min(bin_y, num_bins_y - 1));

    // Linearize the bin index
    int bin_index = bin_y * num_bins_x + bin_x;

    // Store the bin index for this particle
    bin_indices[tid] = bin_index;

    // Atomically increment the bin count (thread-safe)
    atomicAdd(&bin_counts[bin_index], 1);
}

// Device function to apply force between two particles
__device__ void apply_force_gpu(particle_t& particle, particle_t& neighbor) {
    double dx = neighbor.x - particle.x;
    double dy = neighbor.y - particle.y;
    double r2 = dx * dx + dy * dy;

    if (r2 > cutoff * cutoff) return;

    r2 = (r2 > min_r * min_r) ? r2 : min_r * min_r;
    double r = sqrt(r2);

    double coef = (1 - cutoff / r) / r2 / mass;
    atomicAdd(&particle.ax, coef * dx); // Atomic update for thread safety
    atomicAdd(&particle.ay, coef * dy); // Atomic update for thread safety
}

// Kernel to reorder particle indices based on bin assignment
__global__ void reorder_particles_gpu(int* particle_bins, int* bin_indices, int* bin_scan, int num_parts) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= num_parts) return;

    int bin_index = bin_indices[tid];
    int offset = atomicAdd(&bin_scan[bin_index], 1); // Atomic increment to get unique offset
    particle_bins[offset] = tid; // Store particle index
}

// Kernel to compute forces between particles, using binning
__global__ void compute_forces_gpu(particle_t* particles, int num_parts, int* bin_indices, int* bin_scan, int* particle_bins) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= num_parts) return;

    // Reset accelerations
    particles[tid].ax = 0.0;
    particles[tid].ay = 0.0;

    int bin_index = bin_indices[tid];

    // Calculate start and end indices for the current bin (using exclusive prefix sum)
    int bin_start = (bin_index == 0) ? 0 : bin_scan[bin_index - 1];
    int bin_end = bin_scan[bin_index];

    // Iterate over particles within the current bin
    for (int i = bin_start; i < bin_end; ++i) {
        int other_particle_index = particle_bins[i];
        if (tid != other_particle_index) {
            apply_force_gpu(particles[tid], particles[other_particle_index]);
        }
    }

    // Calculate 2D bin coordinates from the linear bin index
    int bin_x = bin_index % num_bins_x;
    int bin_y = bin_index / num_bins_x;

    // Iterate over neighboring bins (including the current bin, handled above)
    for (int dx = -1; dx <= 1; ++dx) {
        for (int dy = -1; dy <= 1; ++dy) {
            if (dx == 0 && dy == 0) continue; // Skip the current bin itself

            int neighbor_bin_x = bin_x + dx;
            int neighbor_bin_y = bin_y + dy;

            // Check if neighboring bin indices are within bounds
            if (neighbor_bin_x >= 0 && neighbor_bin_x < num_bins_x &&
                neighbor_bin_y >= 0 && neighbor_bin_y < num_bins_y) {

                int neighbor_bin_index = neighbor_bin_y * num_bins_x + neighbor_bin_x;

                // Calculate start and end indices for the neighboring bin
                int neighbor_bin_start = (neighbor_bin_index == 0) ? 0 : bin_scan[neighbor_bin_index - 1];
                int neighbor_bin_end = bin_scan[neighbor_bin_index];

                // Iterate over particles in the neighboring bin
                for (int i = neighbor_bin_start; i < neighbor_bin_end; ++i) {
                    int other_particle_index = particle_bins[i];
                    apply_force_gpu(particles[tid], particles[other_particle_index]);
                }
            }
        }
    }
}

// Kernel to move particles based on calculated forces
__global__ void move_gpu(particle_t* particles, int num_parts, double size) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= num_parts) return;

    particle_t* p = &particles[tid];

    // Velocity Verlet integration
    p->vx += p->ax * dt;
    p->vy += p->ay * dt;
    p->x += p->vx * dt;
    p->y += p->vy * dt;

    // Bounce off walls
    while (p->x < 0 || p->x > size) {
        p->x = p->x < 0 ? -p->x : 2 * size - p->x;
        p->vx = -p->vx;
    }
    while (p->y < 0 || p->y > size) {
        p->y = p->y < 0 ? -p->y : 2 * size - p->y;
        p->vy = -p->vy;
    }
}

// Initialization function (host-side)
void init_simulation(particle_t* parts, int num_parts, double size) {
    // calculate the number of blocks
    blks = (num_parts + NUM_THREADS - 1) / NUM_THREADS;

    // calculate the bin parameters
    double host_bin_size = cutoff;
    int host_num_bins_x = static_cast<int>(size / host_bin_size) + 1;
    int host_num_bins_y = static_cast<int>(size / host_bin_size) + 1;
    int num_bins = host_num_bins_x * host_num_bins_y;

    // copy to device
    hipMemcpyToSymbol(HIP_SYMBOL(bin_size), &host_bin_size, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(num_bins_x), &host_num_bins_x, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(num_bins_y), &host_num_bins_y, sizeof(int));

    // release the previous memory (if exists)
    if (d_bin_indices) hipFree(d_bin_indices);
    if (d_bin_counts) hipFree(d_bin_counts);
    if (d_bin_scan) hipFree(d_bin_scan);
    if (d_particle_bins) hipFree(d_particle_bins);

    // allocate new memory
    hipMalloc(&d_bin_indices, num_parts * sizeof(int));
    hipMalloc(&d_bin_counts, num_bins * sizeof(int));
    hipMalloc(&d_bin_scan, num_bins * sizeof(int));
    hipMalloc(&d_particle_bins, num_parts * sizeof(int));
}

// Simulation step function (host-side)
void simulate_one_step(particle_t* parts, int num_parts, double size) {
    int host_num_bins_x, host_num_bins_y;
    hipMemcpyFromSymbol(&host_num_bins_x, HIP_SYMBOL(num_bins_x), sizeof(int));
    hipMemcpyFromSymbol(&host_num_bins_y, HIP_SYMBOL(num_bins_y), sizeof(int));
    int num_bins = host_num_bins_x * host_num_bins_y;

    // reset the bin counts
    hipMemset(d_bin_counts, 0, num_bins * sizeof(int));

    // Step 1: assign particles to bins
    assign_bins_gpu<<<blks, NUM_THREADS>>>(parts, num_parts, d_bin_indices, d_bin_counts);

    // Step 2: Calculate the prefix
    thrust::exclusive_scan(thrust::device_ptr<int>(d_bin_counts),
                           thrust::device_ptr<int>(d_bin_counts + num_bins),
                           thrust::device_ptr<int>(d_bin_scan));

    // Step 3: Reorder the particles
    reorder_particles_gpu<<<blks, NUM_THREADS>>>(d_particle_bins, d_bin_indices, d_bin_scan, num_parts);

    // Step 4: Calculate the forces
    compute_forces_gpu<<<blks, NUM_THREADS>>>(parts, num_parts, d_bin_indices, d_bin_scan, d_particle_bins);

    // Step 5: Move particles
    move_gpu<<<blks, NUM_THREADS>>>(parts, num_parts, size);
}
