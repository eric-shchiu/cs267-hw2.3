#include "hip/hip_runtime.h"
#include "common.h"
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/scan.h>
#include "globals.h"

#define NUM_THREADS 256

// Global variable (host-side) for the number of blocks
int blks;

// __device__ variables (accessible by all kernels)
__device__ double bin_size;
__device__ int num_bins_x, num_bins_y;

int* d_bin_indices = nullptr;
int* d_bin_counts = nullptr;
int* d_bin_scan = nullptr;
int* d_particle_bins = nullptr;

static int host_num_bins_x_cache = 0;
static int host_num_bins_y_cache = 0;

// Kernel to assign particles to bins and count particles per bin
__global__ void assign_bins_gpu(particle_t* particles, int num_parts, int* bin_indices, int* bin_counts) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= num_parts) return;

    // Calculate bin coordinates using __device__ variables
    int bin_x = static_cast<int>(particles[tid].x / bin_size);
    int bin_y = static_cast<int>(particles[tid].y / bin_size);

    // Clamp bin indices to the valid range
    bin_x = max(0, min(bin_x, num_bins_x - 1));
    bin_y = max(0, min(bin_y, num_bins_y - 1));

    // Linearize the bin index
    int bin_index = bin_y * num_bins_x + bin_x;

    // Store the bin index for this particle
    bin_indices[tid] = bin_index;

    // Atomically increment the bin count (thread-safe)
    atomicAdd(&bin_counts[bin_index], 1);
}

// Device function to apply force between two particles
__device__ void apply_force_gpu(particle_t& particle, particle_t& neighbor) {
    double dx = neighbor.x - particle.x;
    double dy = neighbor.y - particle.y;
    double r2 = dx * dx + dy * dy;

    if (r2 > cutoff * cutoff) return;

    r2 = (r2 > min_r * min_r) ? r2 : min_r * min_r;
    double r = sqrt(r2);

    double coef = (1 - cutoff / r) / r2 / mass;
    atomicAdd(&particle.ax, coef * dx); // Atomic update for thread safety
    atomicAdd(&particle.ay, coef * dy); // Atomic update for thread safety
}

// Kernel to reorder particle indices based on bin assignment
__global__ void reorder_particles_gpu(int* particle_bins, int* bin_indices, int* bin_scan, int num_parts) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= num_parts) return;

    int bin_index = bin_indices[tid];
    int offset = atomicAdd(&bin_scan[bin_index], 1); // Atomic increment to get unique offset
    particle_bins[offset] = tid; // Store particle index
}

__device__ void apply_symmetric_force(particle_t& p1, particle_t& p2) {
    double dx = p2.x - p1.x;
    double dy = p2.y - p1.y;
    double r2 = dx*dx + dy*dy;

    if (r2 > cutoff*cutoff) return;

    r2 = fmax(r2, min_r*min_r);
    double r = sqrt(r2);
    double coef = (1 - cutoff/r) / (r2 * mass);

    // Update the acceleration of both particles
    double ax = coef * dx;
    double ay = coef * dy;

    // apply force on both particles
    atomicAdd(&p1.ax, ax);
    atomicAdd(&p1.ay, ay);
    atomicAdd(&p2.ax, -ax);
    atomicAdd(&p2.ay, -ay);
}

__global__ void compute_forces_gpu(particle_t* particles, int num_parts, int* bin_indices, int* bin_scan, int* particle_bins) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= num_parts) return;

    // Reset accelerations
    particles[tid].ax = 0.0;
    particles[tid].ay = 0.0;

    int bin_index = bin_indices[tid];

    // Calculate start and end indices for the current bin (using exclusive prefix sum)
    int bin_start = (bin_index == 0) ? 0 : bin_scan[bin_index - 1];
    int bin_end = bin_scan[bin_index];

    // Iterate over particles within the current bin
    for (int i = bin_start; i < bin_end; ++i) {
        int other_particle_index = particle_bins[i];
        if (tid != other_particle_index) {
            apply_force_gpu(particles[tid], particles[other_particle_index]);
        }
    }

    // Calculate 2D bin coordinates from the linear bin index
    int bin_x = bin_index % num_bins_x;
    int bin_y = bin_index / num_bins_x;

    // Iterate over neighboring bins (right / down / right-down / left-down)
    const int neighbor_dirs[4][2] = {{1,0},  {0,1},  {1,1}, {-1,1}};

    for (int dir = 0; dir < 4; ++dir) {
        int dx = neighbor_dirs[dir][0];
        int dy = neighbor_dirs[dir][1];

        int neighbor_bin_x = bin_x + dx;
        int neighbor_bin_y = bin_y + dy;

        // Check if neighboring bin indices are within bounds
        if (neighbor_bin_x >= 0 && neighbor_bin_x < num_bins_x &&
            neighbor_bin_y >= 0 && neighbor_bin_y < num_bins_y) {

            int neighbor_bin_index = neighbor_bin_y * num_bins_x + neighbor_bin_x;

            // Calculate start and end indices for the neighboring bin
            int neighbor_bin_start = (neighbor_bin_index == 0) ? 0 : bin_scan[neighbor_bin_index - 1];
            int neighbor_bin_end = bin_scan[neighbor_bin_index];

            // Iterate over particles in the neighboring bin
            for (int i = neighbor_bin_start; i < neighbor_bin_end; ++i) {
                int other_particle_index = particle_bins[i];
                apply_symmetric_force(particles[tid], particles[other_particle_index]);
            }
        }
    }
}

// Kernel to move particles based on calculated forces
__global__ void move_gpu(particle_t* particles, int num_parts, double size) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= num_parts) return;

    particle_t* p = &particles[tid];

    // Velocity Verlet integration
    p->vx += p->ax * dt;
    p->vy += p->ay * dt;
    p->x += p->vx * dt;
    p->y += p->vy * dt;

    // Bounce off walls
    while (p->x < 0 || p->x > size) {
        p->x = p->x < 0 ? -p->x : 2 * size - p->x;
        p->vx = -p->vx;
    }
    while (p->y < 0 || p->y > size) {
        p->y = p->y < 0 ? -p->y : 2 * size - p->y;
        p->vy = -p->vy;
    }
}

// Initialization function (host-side)
void init_simulation(particle_t* parts, int num_parts, double size) {
    // calculate the number of blocks
    blks = (num_parts + NUM_THREADS - 1) / NUM_THREADS;

    // calculate the bin parameters
    double host_bin_size = cutoff;
    int host_num_bins_x = static_cast<int>(size / host_bin_size) + 1;
    int host_num_bins_y = static_cast<int>(size / host_bin_size) + 1;

    // Cache calculation results to host variables
    host_num_bins_x_cache = host_num_bins_x;
    host_num_bins_y_cache = host_num_bins_y;

    // calculate the total number of bins
    const int num_bins = host_num_bins_x * host_num_bins_y;

    // copy to device
    hipMemcpyToSymbol(HIP_SYMBOL(bin_size), &host_bin_size, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(num_bins_x), &host_num_bins_x, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(num_bins_y), &host_num_bins_y, sizeof(int));

    // allocate new memory
    hipMalloc(&d_bin_indices, num_parts * sizeof(int));
    hipMalloc(&d_bin_counts, num_bins * sizeof(int));
    hipMalloc(&d_bin_scan, num_bins * sizeof(int));
    hipMalloc(&d_particle_bins, num_parts * sizeof(int));
}

void simulate_one_step(particle_t* parts, int num_parts, double size) {
    const int num_bins = host_num_bins_x_cache * host_num_bins_y_cache;

    // create cuda event
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float computation_time = 0.0f, synchronization_time = 0.0f;

    // Step 1: Reset the bin counts
    hipMemset(d_bin_counts, 0, num_bins * sizeof(int));

    // Assign Bins
    hipEventRecord(start);
    assign_bins_gpu<<<blks, NUM_THREADS>>>(parts, num_parts, d_bin_indices, d_bin_counts);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&computation_time, start, stop);
    comp_time += computation_time / 1000.0; // transfer to seconds

    hipEventRecord(start);
    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&synchronization_time, start, stop);
    sync_time += synchronization_time / 1000.0;

    // Prefix Sum
    hipEventRecord(start);
    thrust::exclusive_scan(thrust::device_ptr<int>(d_bin_counts),
                           thrust::device_ptr<int>(d_bin_counts + num_bins),
                           thrust::device_ptr<int>(d_bin_scan));
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&computation_time, start, stop);
    comp_time += computation_time / 1000.0;

    hipEventRecord(start);
    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&synchronization_time, start, stop);
    sync_time += synchronization_time / 1000.0;

    // Reorder Particles
    hipEventRecord(start);
    reorder_particles_gpu<<<blks, NUM_THREADS>>>(d_particle_bins, d_bin_indices, d_bin_scan, num_parts);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&computation_time, start, stop);
    comp_time += computation_time / 1000.0;

    hipEventRecord(start);
    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&synchronization_time, start, stop);
    sync_time += synchronization_time / 1000.0;

    // Compute Forces
    hipEventRecord(start);
    compute_forces_gpu<<<blks, NUM_THREADS>>>(parts, num_parts, d_bin_indices, d_bin_scan, d_particle_bins);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&computation_time, start, stop);
    comp_time += computation_time / 1000.0;

    hipEventRecord(start);
    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&synchronization_time, start, stop);
    sync_time += synchronization_time / 1000.0;

    // Move Particles
    hipEventRecord(start);
    move_gpu<<<blks, NUM_THREADS>>>(parts, num_parts, size);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&computation_time, start, stop);
    comp_time += computation_time / 1000.0;

    hipEventRecord(start);
    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&synchronization_time, start, stop);
    sync_time += synchronization_time / 1000.0;

    // release cuda event
    hipEventDestroy(start);
    hipEventDestroy(stop);
}
