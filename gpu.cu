#include "hip/hip_runtime.h"
#include "common.h"
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/scan.h>

#define NUM_THREADS 256

// Global variables
int blks;
__device__ double bin_size;
__device__ int num_bins_x, num_bins_y;

int* d_bin_indices;
int* d_bin_counts;
int* d_bin_scan;
int* d_particle_bins;
int* d_bin_offsets;

__device__ void apply_force_gpu(particle_t& particle, particle_t& neighbor) {
    double dx = neighbor.x - particle.x;
    double dy = neighbor.y - particle.y;
    double r2 = dx * dx + dy * dy;
    if (r2 > cutoff * cutoff)
        return;
    // r2 = fmax( r2, min_r*min_r );
    r2 = (r2 > min_r * min_r) ? r2 : min_r * min_r;
    double r = sqrt(r2);

    //
    //  very simple short-range repulsive force
    //
    double coef = (1 - cutoff / r) / r2 / mass;
    particle.ax += coef * dx;
    particle.ay += coef * dy;
}

// Kernel to assign particles to bins, count particles, and reorder them in a single pass
__global__ void bin_particles_gpu(particle_t* particles, int num_parts, int* bin_indices, int* bin_counts, int* bin_offsets) { // 移除非必要参数
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= num_parts) return;

    int bin_x = static_cast<int>(particles[tid].x / bin_size);
    int bin_y = static_cast<int>(particles[tid].y / bin_size);
    bin_x = max(0, min(bin_x, num_bins_x - 1));
    bin_y = max(0, min(bin_y, num_bins_y - 1));

    int bin_index = bin_y * num_bins_x + bin_x;
    bin_indices[tid] = bin_index;

    // 修改3: 保存offset到独立数组
    int offset = atomicAdd(&bin_counts[bin_index], 1);
    bin_offsets[tid] = offset; // 存储每个粒子的偏移
}

__global__ void reorder_particles_gpu(int* bin_indices, int* bin_offsets, int* bin_scan, int* particle_bins, int num_parts) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= num_parts) return;

    int bin_index = bin_indices[tid];
    int start_idx = bin_scan[bin_index];
    int local_offset = bin_offsets[tid];
    particle_bins[start_idx + local_offset] = tid; // 正确填充排序数组
}

// Kernel to compute forces in parallel
__global__ void compute_forces_gpu(particle_t* particles, int num_parts, int* bin_indices, int* bin_scan, int* particle_bins) {
    __shared__ particle_t shared_particles[NUM_THREADS];

    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= num_parts) return;

    particles[tid].ax = 0.0;
    particles[tid].ay = 0.0;

    int bin_index = bin_indices[tid];
    int bin_x = bin_index % num_bins_x;
    int bin_y = bin_index / num_bins_x;

    int num_neighbors = 0;

    for (int dx = -1; dx <= 1; ++dx) {
        for (int dy = -1; dy <= 1; ++dy) {
            int neighbor_bin_x = bin_x + dx;
            int neighbor_bin_y = bin_y + dy;

            if (neighbor_bin_x >= 0 && neighbor_bin_x < num_bins_x &&
                neighbor_bin_y >= 0 && neighbor_bin_y < num_bins_y) {
                int neighbor_bin_index = neighbor_bin_y * num_bins_x + neighbor_bin_x;
                int bin_start = bin_scan[neighbor_bin_index];
                int bin_end = bin_start + bin_counts[neighbor_bin_index];

                for (int i = bin_start; i < bin_end; i += NUM_THREADS) {
                    int index = i + threadIdx.x;
                    if (index < bin_end) {
                        shared_particles[threadIdx.x] = particles[particle_bins[index]];
                    }
                    __syncthreads();

                    for (int j = 0; j < min(NUM_THREADS, bin_end - i); ++j) {
                        apply_force_gpu(particles[tid], shared_particles[j]);
                    }
                    __syncthreads();
                }
            }
        }
    }
}

// Kernel to move particles
__global__ void move_gpu(particle_t* particles, int num_parts, double size) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= num_parts) return;

    particle_t* p = &particles[tid];

    p->vx += p->ax * dt;
    p->vy += p->ay * dt;
    p->x += p->vx * dt;
    p->y += p->vy * dt;

    while (p->x < 0 || p->x > size) {
        p->x = p->x < 0 ? -p->x : 2 * size - p->x;
        p->vx = -p->vx;
    }
    while (p->y < 0 || p->y > size) {
        p->y = p->y < 0 ? -p->y : 2 * size - p->y;
        p->vy = -p->vy;
    }
}

// Initialization function
void init_simulation(particle_t* parts, int num_parts, double size) {
    blks = (num_parts + NUM_THREADS - 1) / NUM_THREADS;
    double host_bin_size = cutoff;
    int host_num_bins_x = static_cast<int>(size / host_bin_size) + 1;
    int host_num_bins_y = static_cast<int>(size / host_bin_size) + 1;

    hipMemcpyToSymbol(HIP_SYMBOL(::bin_size), &host_bin_size, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(::num_bins_x), &host_num_bins_x, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(::num_bins_y), &host_num_bins_y, sizeof(int));

    int num_bins = host_num_bins_x * host_num_bins_y;

    hipMalloc(&d_bin_indices, num_parts * sizeof(int));
    hipMalloc(&d_bin_counts, num_bins * sizeof(int));
    hipMalloc(&d_bin_scan, num_bins * sizeof(int));
    hipMalloc(&d_particle_bins, num_parts * sizeof(int));
    hipMalloc(&d_bin_offsets, num_parts * sizeof(int));
}

// Simulation step function
void simulate_one_step(particle_t* parts, int num_parts, double size) {
    // 获取当前bins配置
    int host_num_bins_x, host_num_bins_y;
    hipMemcpyFromSymbol(&host_num_bins_x, HIP_SYMBOL(::num_bins_x), sizeof(int));
    hipMemcpyFromSymbol(&host_num_bins_y, HIP_SYMBOL(::num_bins_y), sizeof(int));
    const int num_bins = host_num_bins_x * host_num_bins_y;

    // Phase 1: 重置bin计数器并分配粒子到bins
    hipMemset(d_bin_counts, 0, num_bins * sizeof(int));  // 重置计数器
    bin_particles_gpu<<<blks, NUM_THREADS>>>(
        parts, num_parts, 
        d_bin_indices,   // 输出的每个粒子的bin索引
        d_bin_counts,    // 输出的每个bin的粒子计数
        d_bin_offsets    // 输出的每个粒子的bin内偏移
    );
    hipDeviceSynchronize();

    // Phase 2: 计算exclusive前缀和
    thrust::device_ptr<int> thrust_bin_counts(d_bin_counts);
    thrust::device_ptr<int> thrust_bin_scan(d_bin_scan);
    thrust::exclusive_scan(
        thrust_bin_counts, 
        thrust_bin_counts + num_bins, 
        thrust_bin_scan
    );

    // Phase 3: 重排序粒子索引
    reorder_particles_gpu<<<blks, NUM_THREADS>>>(
        d_bin_indices,   // 输入的每个粒子的bin索引
        d_bin_offsets,   // 输入的每个粒子的bin内偏移
        d_bin_scan,      // 输入的bin起始索引
        d_particle_bins, // 输出的按bin排序的粒子索引
        num_parts
    );
    hipDeviceSynchronize();

    // Phase 4: 计算粒子间作用力
    compute_forces_gpu<<<blks, NUM_THREADS>>>(
        parts,          // 粒子数组
        num_parts,      // 粒子总数
        d_bin_indices,  // 每个粒子的bin索引
        d_bin_scan,     // bin起始索引 (exclusive_scan结果)
        d_bin_counts,   // 每个bin的粒子数
        d_particle_bins // 按bin排序的粒子索引
    );
    hipDeviceSynchronize();

    // Phase 5: 移动粒子
    move_gpu<<<blks, NUM_THREADS>>>(parts, num_parts, size);
    hipDeviceSynchronize();
}
